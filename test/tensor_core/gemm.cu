#include <hip/hip_runtime.h>
#include <iostream>
#include <one_tensor.h>
#include "gemm_api_tb256.cuh"
// #include "gemm_api_tb128.cuh"
using namespace std;

#define MEASURE_TIME 1
#define USE_CUBLAS 1
#define DEBUG 0
#define EVAL_TIMES 30
int main()
{

    int measure_times = EVAL_TIMES;
    OneTensor<double> flops(Shape(EVAL_TIMES));
    OneTensor<double> cost_time(Shape(EVAL_TIMES));
    for (int cnt = 1; cnt < measure_times; cnt++)
    {
        float excute_time=0.0;
        
        int M = 16 * 4 * 4 * (cnt + 1);
        int N = 16 * 4 * 4 * (cnt + 1);
        int K=16*4*4*(cnt+1);
        // int K = 16 * 4 * 2 * (1 + 1);
        printf("M: %d\n", M);

        OneTensor<half> a(Shape(M, K));
        OneTensor<half> b(Shape(K, N));
        OneTensor<float> c(Shape(M, N));
        c.FillHostData(0);
        for (int i = 0; i < a.shape.d0; i++)
        {
            for (int j = 0; j < a.shape.d1; j++)
            {
                int index = i * a.shape.d1 + j;
                a.SetHostData(half(1), index);
            }
        }
        for (int i = 0; i < b.shape.d0; i++)
        {
            for (int j = 0; j < b.shape.d1; j++)
            {
                int index = i * b.shape.d1 + j;
                b.SetHostData(half(1), index);
            }
        }

        // a.HostDataView();
        // b.HostDataView();

        a.sync_device();
        b.sync_device();
        c.sync_device();

        dim3 gird(16, div_ceil(M, (WARP_ROW_NUM * MMA_ROW_NUM * MMA_M)), div_ceil(N, (WARP_COL_NUM * MMA_COL_NUM * MMA_N * 16)));
        dim3 block(32 * 8, 1, 1);
        hipStream_t stream;
        hipStreamCreate(&stream);
        if (!(USE_CUBLAS))
        {
            if (!(MEASURE_TIME))
            {
                gemm_fp16fp32_tb256<<<gird, block, 0, stream>>>(a.deviceData<half>(),
                                                                b.deviceData<half>(),
                                                                c.deviceData<float>(),
                                                                M, N, K);
            }
            else
            {
                printf("our: cnt: %d\n", cnt);
                GPU_Time((gemm_fp16fp32_tb256<<<gird, block, 0, stream>>>(a.deviceData<half>(), b.deviceData<half>(), c.deviceData<float>(), M, N, K)),
                         stream,
                         200,
                         0);
                excute_time=mtime;
            }
        }
        else
        {

            float alpha = 1.0;
            float beta = 0.0;
            hipDeviceSynchronize();
            hipblasHandle_t blas_handle;
            hipblasCreate(&blas_handle);
            hipblasSetStream(blas_handle, stream);
            hipblasGemmAlgo_t algo_list[19] = {
                HIPBLAS_GEMM_DEFAULT,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP,
                CUBLAS_GEMM_DFALT_TENSOR_OP,
                CUBLAS_GEMM_ALGO0_TENSOR_OP,
                CUBLAS_GEMM_ALGO1_TENSOR_OP,
                CUBLAS_GEMM_ALGO2_TENSOR_OP,
                CUBLAS_GEMM_ALGO3_TENSOR_OP,
                CUBLAS_GEMM_ALGO4_TENSOR_OP,
                CUBLAS_GEMM_ALGO5_TENSOR_OP,
                CUBLAS_GEMM_ALGO6_TENSOR_OP,
                CUBLAS_GEMM_ALGO7_TENSOR_OP,
                CUBLAS_GEMM_ALGO8_TENSOR_OP,
                CUBLAS_GEMM_ALGO9_TENSOR_OP,
                CUBLAS_GEMM_ALGO10_TENSOR_OP,
                CUBLAS_GEMM_ALGO11_TENSOR_OP,
                CUBLAS_GEMM_ALGO12_TENSOR_OP,
                CUBLAS_GEMM_ALGO13_TENSOR_OP,
                CUBLAS_GEMM_ALGO14_TENSOR_OP,
                CUBLAS_GEMM_ALGO15_TENSOR_OP};
            if (!(MEASURE_TIME))
            {
                (hipblasGemmEx(
                    blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K,
                    &alpha, b.deviceData<half>(), HIP_R_16F, N, a.deviceData<half>(), HIP_R_16F, K, &beta, c.deviceData<float>(), HIP_R_32F, N,
                    HIPBLAS_COMPUTE_32F, algo_list[1]));
            }
            else
            {
                printf("cublas: cnt: %d\n", cnt);
                GPU_Time((hipblasGemmEx(
                             blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K,
                             &alpha, b.deviceData<half>(), HIP_R_16F, N, a.deviceData<half>(), HIP_R_16F, K, &beta, c.deviceData<float>(), HIP_R_32F, N,
                             HIPBLAS_COMPUTE_32F, algo_list[1])),
                         stream, 200, 0);
                excute_time=mtime;

            }
            hipblasDestroy(blas_handle);
        }
        if(DEBUG){
            c.sync_device(false);
            c.HostDataView();
        }

        if (MEASURE_TIME)
        {
            cost_time.SetHostData((excute_time / (200)), cnt - 1);
            flops.SetHostData(M, cnt - 1);
        }
        hipStreamDestroy(stream);
    }

    if (MEASURE_TIME)
    {
        if (USE_CUBLAS)
        {   
            printf("save cublas cost time.\n");
            cost_time.SaveNpyFile<double>("cublas_cost_time.npy");
            flops.SaveNpyFile<double>("cublas_flops.npy");
        }
        else
        {
            printf("save our cost time.\n");
            cost_time.SaveNpyFile<double>("v0_cost_time.npy");
            flops.SaveNpyFile<double>("v0_flops.npy");
        }
    }

    return 0;
}