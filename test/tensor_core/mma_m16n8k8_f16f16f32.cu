#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <one_tensor.h>
using namespace std;

__global__ void mma_m16n8k8(half  *a, 
                            half  *b,
                            float *c,
                            float *d){
    uint32_t *ia0 =(uint32_t*) a;
    uint32_t *ia1 = (uint32_t*)(&a[8*8]);
    uint32_t *ib = (uint32_t*)b;
    float* c0 = c;
    float* c1 =(float*)(&c[8*8]);
    float* d0 = d;
    float* d1 = (float*)(&d[8*8]);

    int tid = blockDim.x*blockIdx.x+threadIdx.x; // wrap: 32Thread.
    int row = tid / 4;
    int col = tid % 4;

    int ab_index = row*4+col;
    int cd_index = row*8+col*2;
    asm volatile(
     "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32 "
     "  {%0,%1,%2,%3},  "
     "  {%4,%5}, "
     "  {%6}, "
     "  {%7,%8,%9,%10};  "
     :
     "=f"(d0[cd_index]),"=f"(d0[cd_index+1]),"=f"(d1[cd_index]),"=f"(d1[cd_index+1])
     :
     "r"(ia0[ab_index]),"r"(ia1[ab_index]),
     "r"(ib[ab_index]),
     "f"(c0[cd_index]),"f"(c0[cd_index+1]),"f"(c1[cd_index]),"f"(c1[cd_index+1])
    );
}

int main(){
    OneTensor<half> a(Shape(16,8));
    OneTensor<half> b(Shape(8,8));
    OneTensor<float> c(Shape(16,8));
    OneTensor<float> d(Shape(16,8));
    c.FillHostData(0);
    d.FillHostData(0);
    for(int i=0; i<a.shape.d0; i++){
        for(int j=0; j<a.shape.d1; j++){
            int index = i*a.shape.d1+j;
            a.SetHostData(j*0.1,index);
            // b.SetHostData(j*1.0,index);
        }
    }
    for(int i=0; i<b.shape.d0; i++){
        for(int j=0; j<b.shape.d1; j++){
            int index = i*b.shape.d1+j;
            b.SetHostData(j*0.1,index);
            // b.SetHostData(j*1.0,index);
        }
    }
    a.HostDataView();
    b.HostDataView();
    // c.HostDataView();
    // d.HostDataView();

    a.sync_device();
    b.sync_device();
    c.sync_device();
    d.sync_device();
   
    dim3 gird(1,1,1);
    dim3 block(32,1,1);
    hipStream_t stream;
    hipStreamCreate(&stream);
    GPU_Time((mma_m16n8k8<<<gird,block,0,stream>>>(a.deviceData<half>(),
                                       b.deviceData<half>(),
                                       c.deviceData<float>(),
                                       d.deviceData<float>()
                                       )), stream,100, 16*8*8);
    hipStreamDestroy(stream);

    d.sync_device(false);
    d.HostDataView();
    int sum=0;
    for(int i=0;i<8;i++){
        sum+=i*i;
    }
    cout<<sum<<endl;

    
    
    return 0;
}