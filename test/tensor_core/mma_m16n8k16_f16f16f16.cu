#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <one_tensor.h>
using namespace std;

__global__ void mma_m16n8k16(half  *a, 
                            half *b,
                            half *c,
                            half *d){
    uint32_t *ia0 =(uint32_t*)a;
    
    uint32_t *ib0 = (uint32_t*)b;
   
    uint32_t* c0 = (uint32_t*)c;
    uint32_t* d0 = (uint32_t*)d;
    

    int tid = blockDim.x*blockIdx.x+threadIdx.x; // wrap: 32Thread.
    int row = tid / 4;
    int col = tid % 4;

    // int ab_index = row*4+col;
    int a0_index = row*8+col;
    int a1_index = 64+row*8+col;
    int a2_index = row*8+col+4;
    int a3_index = 64+row*8+col+4;

    int b0_index = row*8+col;
    int b1_index = row*8+col+4;

    int cd0_index = row*4+col*1;
    int cd1_index = 32+row*4+col*1;

    asm volatile(
     "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16 "
     "  {%0,%1},  "
     "  {%2,%3,%4,%5}, "
     "  {%6,%7}, "
     "  {%8,%9};  "
     :
     "=r"(d0[cd0_index]),"=r"(d0[cd1_index])
     :
     "r"(ia0[a0_index]),"r"(ia0[a1_index]),"r"(ia0[a2_index]),"r"(ia0[a3_index]),
     "r"(ib0[b0_index]),"r"(ib0[b1_index]),
     "r"(c0[cd0_index]),"r"(c0[cd1_index])
    );
}

int main(){
    OneTensor<half> a(Shape(16,16));
    OneTensor<half> b(Shape(8,16));
    OneTensor<half> c(Shape(16,8));
    OneTensor<half> d(Shape(16,8));
    c.FillHostData(half(0));
    d.FillHostData(half(0));
    for(int i=0; i<a.shape.d0; i++){
        for(int j=0; j<a.shape.d1; j++){
            int index = i*a.shape.d1+j;
            a.SetHostData(half(j),index);
            // b.SetHostData(j*1.0,index);
        }
    }
    for(int i=0; i<b.shape.d0; i++){
        for(int j=0; j<b.shape.d1; j++){
            int index = i*b.shape.d1+j;
            b.SetHostData(half(j),index);
            // b.SetHostData(j*1.0,index);
        }
    }
    a.HostDataView();
    b.HostDataView();
    // c.HostDataView();
    // d.HostDataView();

    a.sync_device();
    b.sync_device();
    c.sync_device();
    d.sync_device();
   
    dim3 gird(1,1,1);
    dim3 block(32,1,1);
    hipStream_t stream;
    hipStreamCreate(&stream);
    GPU_Time((mma_m16n8k16<<<gird,block,0,stream>>>(a.deviceData<half>(),
                                       b.deviceData<half>(),
                                       c.deviceData<half>(),
                                       d.deviceData<half>()
                                       )), stream,100,
                                       16*8*16);
    hipStreamDestroy(stream);

    d.sync_device(false);
    d.HostDataView();
    int sum=0;
    for(int i=0;i<16;i++){
        sum+=i*i;
    }
    cout<<sum<<endl;

    
    
    return 0;
}